#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "./common/book.h"
#include "./common/cpu_bitmap.h"

#define INF 2e10f
#define DIM 1024
#define rnd(x) (x * rand() / RAND_MAX)
#define SPHERES (100)


struct Sphere {
    float r, b, g;
    float radius;
    float x, y, z;
    __device__ float hit (float ox, float oy, float *n) {
        float dx = ox - x;
        float dy = oy - y;
        if ((dx * dx + dy * dy) < (radius*radius)) {
            float dz = sqrtf (radius*radius - dx*dx - dy*dy);
            *n = dz / sqrtf (radius * radius);
            return dz + z;
        }
        return -INF;
    }
};


__constant__ Sphere s[SPHERES];

__global__ void kernel (unsigned char *ptr)
{
    // Pixel mapping from threadIdx/BlockIdx
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset  = x + y * blockDim.x * gridDim.x;

    float ox = (x - DIM/2);
    float oy = (y - DIM/2);

    float r = 0, g = 0, b = 0;
    float maxz = -INF;
    for (int i = 0; i < SPHERES; i++) {
        float n;
        float t = s[i].hit (ox, oy, &n);
        if (t > maxz) {
            float fscale = n;
            r = s[i].r * fscale;
            g = s[i].g * fscale;
            b = s[i].b * fscale;
            maxz = t;
        }
    }

    ptr[offset * 4 + 0] = (int) (r * 255);
    ptr[offset * 4 + 1] = (int) (g * 255);
    ptr[offset * 4 + 2] = (int) (b * 255);
    ptr[offset * 4 + 3] = 255;
}


struct DataBlock {
    unsigned char *dev_bitmap;
    Sphere *s;
};


int main (void)
{
    DataBlock data;
    hipEvent_t start, stop;
    HANDLE_ERROR (hipEventCreate (&start));
    HANDLE_ERROR (hipEventCreate (&stop));
    HANDLE_ERROR (hipEventRecord (start, 0));

    CPUBitmap bitmap (DIM, DIM, &data);
    unsigned char *dev_bitmap;

    // Memory allocation in GPU side
    HANDLE_ERROR (hipMalloc ((void **)&dev_bitmap,
                              bitmap.image_size()));

    // Allocate temporary memory, format, Copy into GPU memory, and free
    Sphere *temp_s = (Sphere *)malloc(sizeof (Sphere) * SPHERES);
    for (int i = 0; i < SPHERES; i++) {
        temp_s[i].r = rnd(1.0f);
        temp_s[i].g = rnd(1.0f);
        temp_s[i].b = rnd(1.0f);
        temp_s[i].x = rnd(1000.0f) - 500;
        temp_s[i].y = rnd(1000.0f) - 500;
        temp_s[i].z = rnd(1000.0f) - 500;
        temp_s[i].radius = rnd (100.0f) + 20;
    }

    HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(s), temp_s,
                                      sizeof (Sphere) * SPHERES));
    free (temp_s);

    dim3 grids(DIM/16, DIM/16);
    dim3 threads(16, 16);
    kernel<<<grids, threads>>>(dev_bitmap);

    // Copy Bitmap from GPU
    HANDLE_ERROR (hipMemcpy (bitmap.get_ptr(),
                              dev_bitmap,
                              bitmap.image_size(),
                              hipMemcpyDeviceToHost));


    HANDLE_ERROR (hipEventRecord (stop, 0));
    HANDLE_ERROR (hipEventSynchronize (stop));

    float elapsedTime;
    HANDLE_ERROR (hipEventElapsedTime (&elapsedTime,
                                        start,
                                        stop));
    printf ("Time to generate: %3.1f ms\n", elapsedTime);

    HANDLE_ERROR (hipEventDestroy (start));
    HANDLE_ERROR (hipEventDestroy (stop));

    // Free memory
    HANDLE_ERROR (hipFree (dev_bitmap));

    // Dispay bitmap
    bitmap.display_and_exit ();

    return 0;
}
