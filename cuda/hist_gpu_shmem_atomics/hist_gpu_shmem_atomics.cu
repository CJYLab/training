#include "hip/hip_runtime.h"
#include "./common/book.h"

#define SIZE (100*1024*1024)

__global__ void histo_kernel (unsigned char *buffer,
                              long size,
                              unsigned int *histo)
{
    __shared__ unsigned int temp[256];
    temp[threadIdx.x] = 0;
    __syncthreads ();

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (i < size) {
        atomicAdd (&temp[buffer[i]], 1);
        i += stride;
    }

    __syncthreads ();
    atomicAdd (&(histo[threadIdx.x]), temp[threadIdx.x]);
}


int main (void)
{
    unsigned char *buffer = (unsigned char *)big_random_block (SIZE);

    hipEvent_t start, stop;
    HANDLE_ERROR (hipEventCreate (&start));
    HANDLE_ERROR (hipEventCreate (&stop));
    HANDLE_ERROR (hipEventRecord (start, 0));

    unsigned char *dev_buffer;
    unsigned int  *dev_histo;
    HANDLE_ERROR (hipMalloc ((void **)&dev_buffer, SIZE));
    HANDLE_ERROR (hipMemcpy (dev_buffer, buffer, SIZE, hipMemcpyHostToDevice));
    HANDLE_ERROR (hipMalloc ((void **)&dev_histo,
                              256 * sizeof (int)));
    HANDLE_ERROR (hipMemset (dev_histo, 0,
                              256 * sizeof (int)));

    hipDeviceProp_t prop;
    HANDLE_ERROR (hipGetDeviceProperties (&prop, 0));
    int blocks = prop.multiProcessorCount;
    histo_kernel<<<blocks*2, 256>>>(dev_buffer, SIZE, dev_histo);

    unsigned int histo[256];
    HANDLE_ERROR (hipMemcpy (histo,
                              dev_histo,
                              256 * sizeof (int),
                              hipMemcpyDeviceToHost));

    // Display Timer
    HANDLE_ERROR (hipEventRecord (stop, 0));
    HANDLE_ERROR (hipEventSynchronize(stop));
    float elapsedTime;
    HANDLE_ERROR (hipEventElapsedTime (&elapsedTime, start, stop));
    printf ("Time to generate: %3.1f ms\n", elapsedTime);

    long histoCount = 0;
    for (int i = 0; i < 256; i++) {
        histoCount += histo[i];
    }
    printf ("Histogram Sum: %ld\n", histoCount);

    // veryfy CPU and GPU
    for (int i = 0; i < SIZE; i++) {
        histo[buffer[i]]--;
    }
    for (int i = 0; i < 256; i++) {
        if (histo[i] != 0) {
            printf ("Failure at %d!\n", i);
        }
    }

    HANDLE_ERROR (hipEventDestroy (start));
    HANDLE_ERROR (hipEventDestroy (stop));
    HANDLE_ERROR (hipFree (dev_histo));
    HANDLE_ERROR (hipFree (dev_buffer));
    free (buffer);
    return 0;
}
