#include "hip/hip_runtime.h"
#include "./common/book.h"

#define imin(a,b) (a<b?a:b)

const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N+threadsPerBlock-1)/threadsPerBlock);

__global__ void dot (float *a, float *b, float *c)
{
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    // Set Cache
    cache[cacheIndex] = temp;
    // synchronize threads in the block
    __syncthreads ();

    // This code, threadsPerBlock should be power of 2
    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex+i];
        }
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}

int main (void)
{
    float *a, *b, c, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;

    // allocate memory in CPU
    a = (float *)malloc(N * sizeof (float));
    b = (float *)malloc(N * sizeof (float));
    partial_c = (float *)malloc(blocksPerGrid * sizeof (float));

    // Allocate GPU's memory
    HANDLE_ERROR (hipMalloc ((void **)&dev_a,
                              N * sizeof (float)));
    HANDLE_ERROR (hipMalloc ((void **)&dev_b,
                              N * sizeof (float)));
    HANDLE_ERROR (hipMalloc ((void **)&dev_partial_c,
                              N * sizeof (float)));

    // Set memory data by Host
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Copy Array a & b into GPU
    HANDLE_ERROR (hipMemcpy (dev_a,
                              a,
                              N * sizeof (float),
                              hipMemcpyHostToDevice));
    HANDLE_ERROR (hipMemcpy (dev_b,
                              b,
                              N * sizeof (float),
                              hipMemcpyHostToDevice));

    dot<<<blocksPerGrid, threadsPerBlock>>> (dev_a,
                                             dev_b,
                                             dev_partial_c);

    // copy c from GPU into CPU
    HANDLE_ERROR (hipMemcpy (partial_c,
                              dev_partial_c,
                              blocksPerGrid * sizeof (float),
                              hipMemcpyDeviceToHost));

    // Final calculation ni CPU
    c = 0;
    for (int i = 0; i < blocksPerGrid; i++) {
        c += partial_c[i];
    }

#define sum_squares(x) (x*(x+1)*(2*x+1)/6)
    printf ("Des GPU value %.6g = %.6g?\n", c,
            2 * sum_squares ((float)(N-1)));

    // Free GPU's memory
    HANDLE_ERROR (hipFree (dev_a));
    HANDLE_ERROR (hipFree (dev_b));
    HANDLE_ERROR (hipFree (dev_partial_c));

    // Free CPU's memory
    free (a);
    free (b);
    free (partial_c);

    return 0;

}
