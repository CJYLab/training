#include "hip/hip_runtime.h"
#include "./common/book.h"

#define N 10


__global__ void add (int *a, int *b, int *c)
{
    int tid = threadIdx.x;

    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}



int main (void)
{
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    // Allocate GPU's memory
    HANDLE_ERROR (hipMalloc ((void **)&dev_a, N * sizeof (int)));
    HANDLE_ERROR (hipMalloc ((void **)&dev_b, N * sizeof (int)));
    HANDLE_ERROR (hipMalloc ((void **)&dev_c, N * sizeof (int)));

    // Set Array a and b by CPU
    for (int i = 0; i < N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    // Copy Array a and b into GPU
    HANDLE_ERROR (hipMemcpy (dev_a, a, N * sizeof (int),
                              hipMemcpyHostToDevice));
    HANDLE_ERROR (hipMemcpy (dev_b, b, N * sizeof (int),
                              hipMemcpyHostToDevice));

    add<<<1, N>>> (dev_a, dev_b, dev_c);

    // Copy Array c into CPU
    HANDLE_ERROR (hipMemcpy (c, dev_c, N * sizeof(int),
                              hipMemcpyDeviceToHost));

    // Display result
    for (int i = 0; i < N; i++) {
        printf ("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // Free GPU's memory
    HANDLE_ERROR (hipFree (dev_a));
    HANDLE_ERROR (hipFree (dev_b));
    HANDLE_ERROR (hipFree (dev_c));

    return 0;
}
