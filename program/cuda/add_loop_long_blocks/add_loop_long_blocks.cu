#include "hip/hip_runtime.h"
#include "./common/book.h"

#define N (33 * 1024)

__global__ void add (int *a, int *b, int *c)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        // tid += blockDim.x * gridDim.x;
        tid += blockDim.x * gridDim.x;
    }
}

int main (void)
{
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;

    // Allocate CPU's memory
    a = (int *) malloc (N * sizeof (int));
    b = (int *) malloc (N * sizeof (int));
    c = (int *) malloc (N * sizeof (int));

    // Allocate GPU's memory
    HANDLE_ERROR (hipMalloc ((void **)&dev_a,
                              N * sizeof (int)));
    HANDLE_ERROR (hipMalloc ((void **)&dev_b,
                              N * sizeof (int)));
    HANDLE_ERROR (hipMalloc ((void **)&dev_c,
                              N * sizeof (int)));

    // Set Array a & b
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = 2 * i;
    }

    // Copy Array a & b into GPU
    HANDLE_ERROR (hipMemcpy (dev_a,
                              a,
                              N * sizeof (int),
                              hipMemcpyHostToDevice));
    HANDLE_ERROR (hipMemcpy (dev_b,
                              b,
                              N * sizeof (int),
                              hipMemcpyHostToDevice));

    add <<<128, 128>>> (dev_a, dev_b, dev_c);

    // Copy Array c into CPU
    HANDLE_ERROR (hipMemcpy (c,
                              dev_c,
                              N * sizeof (int),
                              hipMemcpyDeviceToHost));

    // check GPU did it that requested from CPU
    bool success = true;
    for (int i = 0; i < N; i++) {
        if ((a[i] + b[i]) != c[i]) {
            printf ("Error: %d + %d != %d\n", a[i], b[i], c[i]);
            success = false;
        }
    }
    if (success) printf ("We did it!\n");

    // Free GPU's allocated memory
    HANDLE_ERROR (hipFree (dev_a));
    HANDLE_ERROR (hipFree (dev_b));
    HANDLE_ERROR (hipFree (dev_c));

    // Free CPU's allocated memory
    free (a);
    free (b);
    free (c);

    return 0;
}
